#include "hip/hip_runtime.h"
#include "chelpers_steps_gpu.hpp"

__device__  void CHelpersStepsGPU::storePolinomials_(StarkInfo &starkInfo, StepsParams &params, uint64_t row, uint64_t domainExtended) {

        /*bufferT_ = bufferT_d[blockIdx.x];
        if(domainExtended) {
            // Store either polinomial f or polinomial q
            for(uint64_t k = 0; k < nColsStages[10]; ++k) {
                Goldilocks::Element *buffT = &bufferT_[(nColsStagesAcc[10] + k)* nrowsPack];
                Goldilocks::copy_pack( &pols_d[offsetsStages[10] + k + row * nColsStages[10]], nColsStages[10], buffT);
            }
        } else {
            uint64_t nStages = 3;
            uint64_t domainSize =  1 << starkInfo.starkStruct.nBits;
            for(uint64_t s = 2; s <= nStages + 1; ++s) {
                bool isTmpPol =  s == 4;
                for(uint64_t k = 0; k < nColsStages[s]; ++k) {
                    uint64_t dim = storePol[nColsStagesAcc[s] + k];
                    if(storePol[nColsStagesAcc[s] + k]) {
                        Goldilocks::Element *buffT = &bufferT_[(nColsStagesAcc[s] + k)* nrowsPack];
                        if(isTmpPol) {
                            for(uint64_t i = 0; i < dim; ++i) {
                                Goldilocks::copy_pack(&params.pols[offsetsStages[s] + k * domainSize + row * dim + i], uint64_t(dim), &buffT[i*nrowsPack]);
                            }
                        } else {
                            Goldilocks::copy_pack(&params.pols[offsetsStages[s] + k + row * nColsStages[s]], nColsStages[s], buffT);
                        }
                    }
                }
            }
        }*/
    }



__device__   void CHelpersStepsGPU::loadPolinomials_(StarkInfo &starkInfo, StepsParams &params, uint64_t row, uint64_t stage, uint64_t domainExtended) {
    /*
        // buffered data
        Goldilocks::Element *constPols_aux = domainExtended ? constPols2ns_d : constPols_d;
        uint64_t domainSize = domainExtended ? 1 << starkInfo.starkStruct.nBitsExt : 1 << starkInfo.starkStruct.nBits;
        Goldilocks::Element *x_aux = domainExtended ? x_2ns_d : x_d;

        uint64_t nStages = 3;
        uint64_t nextStride = domainExtended ?  1 << (starkInfo.starkStruct.nBitsExt - starkInfo.starkStruct.nBits) : 1;
        std::vector<uint64_t> nextStrides = {0, nextStride};
        for(uint64_t k = 0; k < starkInfo.nConstants; ++k) {
            for(uint64_t o = 0; o < 2; ++o) {
                uint64_t l = (row + threadIdx.x + nextStrides[o]) % domainSize;
                bufferT_[(nColsStagesAcc[5*o] + k)*nrowsPack + threadIdx.x] = constPols_aux[l * starkInfo.nConstants + k];
            }
        }

        // Load x and Zi
        bufferT_[starkInfo.nConstants*nrowsPack + threadIdx.x] = x_aux[row + threadIdx.x];
        bufferT_[(starkInfo.nConstants + 1)*nrowsPack + threadIdx.x] = zi_d[row + threadIdx.x];
        

        for(uint64_t s = 1; s <= nStages; ++s) {
            for(uint64_t k = 0; k < nColsStages[s]; ++k) {
                for(uint64_t o = 0; o < 2; ++o) {
                        uint64_t l = (row + j + nextStrides[o]) % domainSize;
                        bufferT_[(nColsStagesAcc[5*o + s] + k)*nrowsPack + threadIdx.x] = pols_d[offsetsStages[s] + l * nColsStages[s] + k];
                }
            }
        }

        if(stage == 5) {
            for(uint64_t k = 0; k < nColsStages[nStages + 1]; ++k) {
                for(uint64_t o = 0; o < 2; ++o) {
                    uint64_t l = (row + j + nextStrides[o]) % domainSize;
                    bufferT_[(nColsStagesAcc[5*o + nStages + 1] + k)*nrowsPack + threadIdx.x] = pols_d[offsetsStages[nStages + 1] + l * nColsStages[nStages + 1] + k];
                }
            }

           // Load xDivXSubXi & xDivXSubWXi
           for(uint64_t d = 0; d < 2; ++d) {
               for(uint64_t i = 0; i < FIELD_EXTENSION; ++i) {
                    bufferT_[(nColsStagesAcc[11] + FIELD_EXTENSION*d + i)*nrowsPack + threadIdx.x] = xDivXSubXi_d[(d*domainSize + row + j)*FIELD_EXTENSION+i];SSSSS                   
               }
           }
        }
        */
    }

 void CHelpersStepsGPU::calculateExpressions_(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams)  {
    bool domainExtended = parserParams.stage > 3 ? true : false;
    uint64_t domainSize = domainExtended ? 1 << starkInfo.starkStruct.nBitsExt : 1 << starkInfo.starkStruct.nBits;
    

    /*setBufferTInfo(starkInfo, parserParams.stage);
    dataSetup(starkInfo, params, parserArgs, parserParams);
    blockCalculation<<<numBlocks, 256>>>(starkInfo,params, domainSize, dominExtended, stage, nOps, nArgs);
    */
}

__global__ void blockCalculation(StarkInfo &starkInfo, StepsParams &params, uint64_t domainSize, bool domainExtended, uint64_t stage, uint32_t nOps, uint32_t nArgs) {
    uint32_t i = threadIdx.x;
    /*while( i < domainSize) {
        loadPolinomials(starkInfo, params, i, stage, nrowsPack, domainExtended);
        optcodeIteration(nrowsPack,  nOps, nArgs);
        storePolinomials(starkInfo, params, storePol_d, i, nrowsPack, domainExtended);
        i += blockDim.x;
    }*/
}


    __device__  void CHelpersStepsGPU::optcodeIteration_(uint32_t nOps, uint32_t nArgs) {

    uint64_t i_args = 0;
    Goldilocks::Element *bufferT_ = bufferT_d[blockIdx.x];
    Goldilocks::Element *tmp1 = tmp1_d[blockIdx.x];
    Goldilocks::Element *tmp3 = tmp3_d[blockIdx.x];
#if 0
    for (uint64_t kk = 0; kk < nOps; ++kk) {
        
        switch (ops_d[kk]) {
            case 0: {
                // COPY commit1 to commit1
                Goldilocks::copy_gpu(&bufferT_[(nColsStagesAcc[args_d[i_args]] + args_d[i_args + 1]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x]);
                i_args += 4;
                break;
            }
            case 1: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: commit1
                Goldilocks::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 5]] + args_d[i_args + 6]) * blockDim.x]);
                i_args += 7;
                break;
            }
            case 2: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: tmp1
                Goldilocks::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &tmp1[args_d[i_args + 5] * blockDim.x]);
                i_args += 6;
                break;
            }
            case 3: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: public
                Goldilocks::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &publics_d[args_d[i_args + 5] * blockDim.x]);
                i_args += 6;
                break;
            }
            case 4: {
                // OPERATION WITH DEST: commit1 - SRC0: commit1 - SRC1: number
                Goldilocks::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &numbers_d[args_d[i_args + 5]*blockDim.x]);
                i_args += 6;
                break;
            }
            case 5: {
                // COPY tmp1 to commit1
                Goldilocks::copy_gpu(&bufferT_[(nColsStagesAcc[args_d[i_args]] + args_d[i_args + 1]) * blockDim.x], &tmp1[args_d[i_args + 2] * blockDim.x]);
                i_args += 3;
                break;
            }
            case 6: {
                // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: tmp1
                Goldilocks::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &tmp1[args_d[i_args + 3] * blockDim.x], &tmp1[args_d[i_args + 4] * blockDim.x]);
                i_args += 5;
                break;
            }
            case 7: {
                // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: public
                Goldilocks::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &tmp1[args_d[i_args + 3] * blockDim.x], &publics_d[args_d[i_args + 4] * blockDim.x]);
                i_args += 5;
                break;
            }
            case 8: {
                // OPERATION WITH DEST: commit1 - SRC0: tmp1 - SRC1: number
                Goldilocks::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &tmp1[args_d[i_args + 3] * blockDim.x], &numbers_d[args_d[i_args + 4]*blockDim.x]);
                i_args += 5;
                break;
            }
            case 9: {
                // COPY public to commit1
                Goldilocks::copy_gpu(&bufferT_[(nColsStagesAcc[args_d[i_args]] + args_d[i_args + 1]) * blockDim.x], &publics_d[args_d[i_args + 2] * blockDim.x]);
                i_args += 3;
                break;
            }
            case 10: {
                // OPERATION WITH DEST: commit1 - SRC0: public - SRC1: public
                Goldilocks::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &publics_d[args_d[i_args + 3] * blockDim.x], &publics_d[args_d[i_args + 4] * blockDim.x]);
                i_args += 5;
                break;
            }
            case 11: {
                // OPERATION WITH DEST: commit1 - SRC0: public - SRC1: number
                Goldilocks::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &publics_d[args_d[i_args + 3] * blockDim.x], &numbers_d[args_d[i_args + 4]*blockDim.x]);
                i_args += 5;
                break;
            }
            case 12: {
                // COPY number to commit1
                Goldilocks::copy_gpu(&bufferT_[(nColsStagesAcc[args_d[i_args]] + args_d[i_args + 1]) * blockDim.x], &numbers_d[args_d[i_args + 2]*blockDim.x]);
                i_args += 3;
                break;
            }
            case 13: {
                // OPERATION WITH DEST: commit1 - SRC0: number - SRC1: number
                Goldilocks::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &numbers_d[args_d[i_args + 3]*blockDim.x], &numbers_d[args_d[i_args + 4]*blockDim.x]);
                i_args += 5;
                break;
            }
            case 14: {
                // COPY commit1 to tmp1
                Goldilocks::copy_gpu(&tmp1[args_d[i_args] * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x]);
                i_args += 3;
                break;
            }
            case 15: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: commit1
                Goldilocks::op_gpu(args_d[i_args], &tmp1[args_d[i_args + 1] * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 4]] + args_d[i_args + 5]) * blockDim.x]);
                i_args += 6;
                break;
            }
            case 16: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: tmp1
                Goldilocks::op_gpu(args_d[i_args], &tmp1[args_d[i_args + 1] * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &tmp1[args_d[i_args + 4] * blockDim.x]);
                i_args += 5;
                break;
            }
            case 17: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: public
                Goldilocks::op_gpu(args_d[i_args], &tmp1[args_d[i_args + 1] * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &publics_d[args_d[i_args + 4] * blockDim.x]);
                i_args += 5;
                break;
            }
            case 18: {
                // OPERATION WITH DEST: tmp1 - SRC0: commit1 - SRC1: number
                Goldilocks::op_gpu(args_d[i_args], &tmp1[args_d[i_args + 1] * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &numbers_d[args_d[i_args + 4]*blockDim.x]);
                i_args += 5;
                break;
            }
            case 19: {
                // COPY tmp1 to tmp1
                Goldilocks::copy_gpu(&tmp1[args_d[i_args] * blockDim.x], &tmp1[args_d[i_args + 1] * blockDim.x]);
                i_args += 2;
                break;
            }
            case 20: {
                // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: tmp1
                Goldilocks::op_gpu(args_d[i_args], &tmp1[args_d[i_args + 1] * blockDim.x], &tmp1[args_d[i_args + 2] * blockDim.x], &tmp1[args_d[i_args + 3] * blockDim.x]);
                i_args += 4;
                break;
            }
            case 21: {
                // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: public
                Goldilocks::op_gpu(args_d[i_args], &tmp1[args_d[i_args + 1] * blockDim.x], &tmp1[args_d[i_args + 2] * blockDim.x], &publics_d[args_d[i_args + 3] * blockDim.x]);
                i_args += 4;
                break;
            }
            case 22: {
                // OPERATION WITH DEST: tmp1 - SRC0: tmp1 - SRC1: number
                Goldilocks::op_gpu(args_d[i_args], &tmp1[args_d[i_args + 1] * blockDim.x], &tmp1[args_d[i_args + 2] * blockDim.x], &numbers_d[args_d[i_args + 3]*blockDim.x]);
                i_args += 4;
                break;
            }
            case 23: {
                // COPY public to tmp1
                Goldilocks::copy_gpu(&tmp1[args_d[i_args] * blockDim.x], &publics_d[args_d[i_args + 1] * blockDim.x]);
                i_args += 2;
                break;
            }
            case 24: {
                // OPERATION WITH DEST: tmp1 - SRC0: public - SRC1: public
                Goldilocks::op_gpu(args_d[i_args], &tmp1[args_d[i_args + 1] * blockDim.x], &publics_d[args_d[i_args + 2] * blockDim.x], &publics_d[args_d[i_args + 3] * blockDim.x]);
                i_args += 4;
                break;
            }
            case 25: {
                // OPERATION WITH DEST: tmp1 - SRC0: public - SRC1: number
                Goldilocks::op_gpu(args_d[i_args], &tmp1[args_d[i_args + 1] * blockDim.x], &publics_d[args_d[i_args + 2] * blockDim.x], &numbers_d[args_d[i_args + 3]*blockDim.x]);
                i_args += 4;
                break;
            }
            case 26: {
                // COPY number to tmp1
                Goldilocks::copy_gpu(&tmp1[args_d[i_args] * blockDim.x], &numbers_d[args_d[i_args + 1]*blockDim.x]);
                i_args += 2;
                break;
            }
            case 27: {
                // OPERATION WITH DEST: tmp1 - SRC0: number - SRC1: number
                Goldilocks::op_gpu(args_d[i_args], &tmp1[args_d[i_args + 1] * blockDim.x], &numbers_d[args_d[i_args + 2]*blockDim.x], &numbers_d[args_d[i_args + 3]*blockDim.x]);
                i_args += 4;
                break;
            }
            case 28: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: commit1
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 5]] + args_d[i_args + 6]) * blockDim.x]);
                i_args += 7;
                break;
            }
            case 29: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: tmp1
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &tmp1[args_d[i_args + 5] * blockDim.x]);
                i_args += 6;
                break;
            }
            case 30: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: public
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &publics_d[args_d[i_args + 5] * blockDim.x]);
                i_args += 6;
                break;
            }
            case 31: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: number
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &numbers_d[args_d[i_args + 5]*blockDim.x]);
                i_args += 6;
                break;
            }
            case 32: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: commit1
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &tmp3[args_d[i_args + 3] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 4]] + args_d[i_args + 5]) * blockDim.x]);
                i_args += 6;
                break;
            }
            case 33: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: tmp1
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &tmp3[args_d[i_args + 3] * blockDim.x * FIELD_EXTENSION], &tmp1[args_d[i_args + 4] * blockDim.x]);
                i_args += 5;
                break;
            }
            case 34: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: public
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &tmp3[args_d[i_args + 3] * blockDim.x * FIELD_EXTENSION], &publics_d[args_d[i_args + 4] * blockDim.x]);
                i_args += 5;
                break;
            }
            case 35: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: number
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &tmp3[args_d[i_args + 3] * blockDim.x * FIELD_EXTENSION], &numbers_d[args_d[i_args + 4]*blockDim.x]);
                i_args += 5;
                break;
            }
            case 36: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: commit1
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &challenges_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 4]] + args_d[i_args + 5]) * blockDim.x]);
                i_args += 6;
                break;
            }
            case 37: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: tmp1
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &challenges_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x], &tmp1[args_d[i_args + 4] * blockDim.x]);
                i_args += 5;
                break;
            }
            case 38: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: public
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &challenges_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x], &publics_d[args_d[i_args + 4] * blockDim.x]);
                i_args += 5;
                break;
            }
            case 39: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: number
                Goldilocks3::op_31_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &challenges_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x], &numbers_d[args_d[i_args + 4]*blockDim.x]);
                i_args += 5;
                break;
            }
            case 40: {
                // COPY commit3 to commit3
                Goldilocks3::copy_gpu(&bufferT_[(nColsStagesAcc[args_d[i_args]] + args_d[i_args + 1]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x]);
                i_args += 4;
                break;
            }
            case 41: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: commit3
                Goldilocks3::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 5]] + args_d[i_args + 6]) * blockDim.x]);
                i_args += 7;
                break;
            }
            case 42: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: tmp3
                Goldilocks3::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &tmp3[args_d[i_args + 5] * blockDim.x * FIELD_EXTENSION]);
                i_args += 6;
                break;
            }
            case 43: {
                // MULTIPLICATION WITH DEST: commit3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3::mul_gpu(&bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &challenges_d[args_d[i_args + 5]*FIELD_EXTENSION*blockDim.x], &challenges_ops_d[args_d[i_args + 5]*FIELD_EXTENSION*blockDim.x]);
                i_args += 6;
                break;
            }
            case 44: {
                // OPERATION WITH DEST: commit3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x], &challenges_d[args_d[i_args + 5]*FIELD_EXTENSION*blockDim.x]);
                i_args += 6;
                break;
            }
            case 45: {
                // COPY tmp3 to commit3
                Goldilocks3::copy_gpu(&bufferT_[(nColsStagesAcc[args_d[i_args]] + args_d[i_args + 1]) * blockDim.x], &tmp3[args_d[i_args + 2] * blockDim.x * FIELD_EXTENSION]);
                i_args += 3;
                break;
            }
            case 46: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: tmp3
                Goldilocks3::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &tmp3[args_d[i_args + 3] * blockDim.x * FIELD_EXTENSION], &tmp3[args_d[i_args + 4] * blockDim.x * FIELD_EXTENSION]);
                i_args += 5;
                break;
            }
            case 47: {
                // MULTIPLICATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3::mul_gpu(&bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &tmp3[args_d[i_args + 3] * blockDim.x * FIELD_EXTENSION], &challenges_d[args_d[i_args + 4]*FIELD_EXTENSION*blockDim.x], &challenges_ops_d[args_d[i_args + 4]*FIELD_EXTENSION*blockDim.x]);
                i_args += 5;
                break;
            }
            case 48: {
                // OPERATION WITH DEST: commit3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &tmp3[args_d[i_args + 3] * blockDim.x * FIELD_EXTENSION], &challenges_d[args_d[i_args + 4]*FIELD_EXTENSION*blockDim.x]);
                i_args += 5;
                break;
            }
            case 49: {
                // MULTIPLICATION WITH DEST: commit3 - SRC0: challenge - SRC1: challenge
                Goldilocks3::mul_gpu(&bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &challenges_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x], &challenges_d[args_d[i_args + 4]*FIELD_EXTENSION*blockDim.x], &challenges_ops_d[args_d[i_args + 4]*FIELD_EXTENSION*blockDim.x]);
                i_args += 5;
                break;
            }
            case 50: {
                // OPERATION WITH DEST: commit3 - SRC0: challenge - SRC1: challenge
                Goldilocks3::op_gpu(args_d[i_args], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x], &challenges_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x], &challenges_d[args_d[i_args + 4]*FIELD_EXTENSION*blockDim.x]);
                i_args += 5;
                break;
            }
            case 51: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: commit1
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 4]] + args_d[i_args + 5]) * blockDim.x]);
                i_args += 6;
                break;
            }
            case 52: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: tmp1
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &tmp1[args_d[i_args + 4] * blockDim.x]);
                i_args += 5;
                break;
            }
            case 53: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: public
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &publics_d[args_d[i_args + 4] * blockDim.x]);
                i_args += 5;
                break;
            }
            case 54: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: number
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &numbers_d[args_d[i_args + 4]*blockDim.x]);
                i_args += 5;
                break;
            }
            case 55: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: commit1
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_d[i_args + 2] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x]);
                i_args += 5;
                break;
            }
            case 56: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: tmp1
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_d[i_args + 2] * blockDim.x * FIELD_EXTENSION], &tmp1[args_d[i_args + 3] * blockDim.x]);
                i_args += 4;
                break;
            }
            case 57: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: public
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_d[i_args + 2] * blockDim.x * FIELD_EXTENSION], &publics_d[args_d[i_args + 3] * blockDim.x]);
                i_args += 4;
                break;
            }
            case 58: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: number
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_d[i_args + 2] * blockDim.x * FIELD_EXTENSION], &numbers_d[args_d[i_args + 3]*blockDim.x]);
                i_args += 4;
                break;
            }
            case 59: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: commit1
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &challenges_d[args_d[i_args + 2]*FIELD_EXTENSION*blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x]);
                i_args += 5;
                break;
            }
            case 60: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: tmp1
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &challenges_d[args_d[i_args + 2]*FIELD_EXTENSION*blockDim.x], &tmp1[args_d[i_args + 3] * blockDim.x]);
                i_args += 4;
                break;
            }
            case 61: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: public
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &challenges_d[args_d[i_args + 2]*FIELD_EXTENSION*blockDim.x], &publics_d[args_d[i_args + 3] * blockDim.x]);
                i_args += 4;
                break;
            }
            case 62: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: number
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &challenges_d[args_d[i_args + 2]*FIELD_EXTENSION*blockDim.x], &numbers_d[args_d[i_args + 3]*blockDim.x]);
                i_args += 4;
                break;
            }
            case 63: {
                // COPY commit3 to tmp3
                Goldilocks3::copy_gpu(&tmp3[args_d[i_args] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 1]] + args_d[i_args + 2]) * blockDim.x]);
                i_args += 3;
                break;
            }
            case 64: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: commit3
                Goldilocks3::op_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 4]] + args_d[i_args + 5]) * blockDim.x]);
                i_args += 6;
                break;
            }
            case 65: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: tmp3
                Goldilocks3::op_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &tmp3[args_d[i_args + 4] * blockDim.x * FIELD_EXTENSION]);
                i_args += 5;
                break;
            }
            case 66: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3::mul_gpu(&tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &challenges_d[args_d[i_args + 4]*FIELD_EXTENSION*blockDim.x], &challenges_ops_d[args_d[i_args + 4]*FIELD_EXTENSION*blockDim.x]);
                i_args += 5;
                break;
            }
            case 67: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: challenge
                Goldilocks3::op_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &challenges_d[args_d[i_args + 4]*FIELD_EXTENSION*blockDim.x]);
                i_args += 5;
                break;
            }
            case 68: {
                // COPY tmp3 to tmp3
                Goldilocks3::copy_gpu(&tmp3[args_d[i_args] * blockDim.x * FIELD_EXTENSION], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION]);
                i_args += 2;
                break;
            }
            case 69: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: tmp3
                Goldilocks3::op_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_d[i_args + 2] * blockDim.x * FIELD_EXTENSION], &tmp3[args_d[i_args + 3] * blockDim.x * FIELD_EXTENSION]);
                i_args += 4;
                break;
            }
            case 70: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3::mul_gpu(&tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_d[i_args + 2] * blockDim.x * FIELD_EXTENSION], &challenges_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x], &challenges_ops_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x]);
                i_args += 4;
                break;
            }
            case 71: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: challenge
                Goldilocks3::op_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_d[i_args + 2] * blockDim.x * FIELD_EXTENSION], &challenges_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x]);
                i_args += 4;
                break;
            }
            case 72: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: challenge - SRC1: challenge
                Goldilocks3::mul_gpu(&tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &challenges_d[args_d[i_args + 2]*FIELD_EXTENSION*blockDim.x], &challenges_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x], &challenges_ops_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x]);
                i_args += 4;
                break;
            }
            case 73: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: challenge
                Goldilocks3::op_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &challenges_d[args_d[i_args + 2]*FIELD_EXTENSION*blockDim.x], &challenges_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x]);
                i_args += 4;
                break;
            }
            case 74: {
                // COPY eval to tmp3
                Goldilocks3::copy_gpu(&tmp3[args_d[i_args] * blockDim.x * FIELD_EXTENSION], &evals_d[args_d[i_args + 1]*FIELD_EXTENSION*blockDim.x]);
                i_args += 2;
                break;
            }
            case 75: {
                // MULTIPLICATION WITH DEST: tmp3 - SRC0: eval - SRC1: challenge
                Goldilocks3::mul_gpu(&tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &evals_d[args_d[i_args + 2]*FIELD_EXTENSION*blockDim.x], &challenges_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x], &challenges_ops_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x]);
                i_args += 4;
                break;
            }
            case 76: {
                // OPERATION WITH DEST: tmp3 - SRC0: challenge - SRC1: eval
                Goldilocks3::op_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &challenges_d[args_d[i_args + 2]*FIELD_EXTENSION*blockDim.x], &evals_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x]);
                i_args += 4;
                break;
            }
            case 77: {
                // OPERATION WITH DEST: tmp3 - SRC0: tmp3 - SRC1: eval
                Goldilocks3::op_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &tmp3[args_d[i_args + 2] * blockDim.x * FIELD_EXTENSION], &evals_d[args_d[i_args + 3]*FIELD_EXTENSION*blockDim.x]);
                i_args += 4;
                break;
            }
            case 78: {
                // OPERATION WITH DEST: tmp3 - SRC0: eval - SRC1: commit1
                Goldilocks3::op_31_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &evals_d[args_d[i_args + 2]*FIELD_EXTENSION*blockDim.x], &bufferT_[(nColsStagesAcc[args_d[i_args + 3]] + args_d[i_args + 4]) * blockDim.x]);
                i_args += 5;
                break;
            }
            case 79: {
                // OPERATION WITH DEST: tmp3 - SRC0: commit3 - SRC1: eval
                Goldilocks3::op_gpu(args_d[i_args], &tmp3[args_d[i_args + 1] * blockDim.x * FIELD_EXTENSION], &bufferT_[(nColsStagesAcc[args_d[i_args + 2]] + args_d[i_args + 3]) * blockDim.x], &evals_d[args_d[i_args + 4]*FIELD_EXTENSION*blockDim.x]);
                i_args += 5;
                break;
            }
                default: {
                    std::cout << " Wrong operation!" << std::endl;
                    exit(1);
                }
            }
        }
        assert(i_args == nArgs);
    #endif
    }
 
void CHelpersStepsGPU::dataSetup_(StarkInfo &starkInfo, StepsParams &params, ParserArgs &parserArgs, ParserParams &parserParams){

        uint32_t nrowsPack =  4;

        /*
            non-buffered data
        */
        uint8_t *ops = &parserArgs.ops[parserParams.opsOffset];
        ops_d = new uint32_t[parserParams.nOps];
        for(uint64_t i = 0; i < parserParams.nOps; ++i) ops_d[i] = uint32_t(ops[i]);
        
        uint16_t *args = &parserArgs.args[parserParams.argsOffset];
        args_d = new uint32_t[parserParams.nArgs];
        for(uint64_t i = 0; i < parserParams.nArgs; ++i) args_d[i] = uint32_t(args[i]);

        uint8_t *storePol = &parserArgs.storePols[parserParams.storePolsOffset];
        storePol_d = new uint32_t[parserParams.nStorePols];
        for(uint64_t i = 0; i < parserParams.nStorePols; ++i) storePol_d[i] = uint32_t(storePol[i]);

        Goldilocks::Element challenges[params.challenges.degree()*FIELD_EXTENSION*nrowsPack];
        Goldilocks::Element challenges_ops[params.challenges.degree()*FIELD_EXTENSION*nrowsPack];
        for(uint64_t i = 0; i < params.challenges.degree(); ++i) {
            for(uint64_t j = 0; j < nrowsPack; ++j) {
                challenges[(i*FIELD_EXTENSION)*nrowsPack + j] = params.challenges[i][0];
                challenges[(i*FIELD_EXTENSION + 1)*nrowsPack + j] = params.challenges[i][1];
                challenges[(i*FIELD_EXTENSION + 2)*nrowsPack + j] = params.challenges[i][2];
                challenges_ops[(i*FIELD_EXTENSION)*nrowsPack + j] = params.challenges[i][0] + params.challenges[i][1];
                challenges_ops[(i*FIELD_EXTENSION + 1)*nrowsPack + j] = params.challenges[i][0] + params.challenges[i][2];
                challenges_ops[(i*FIELD_EXTENSION + 2)*nrowsPack + j] = params.challenges[i][1] + params.challenges[i][2];
            }
        }
        hipMalloc(&challenges_d, params.challenges.degree()*FIELD_EXTENSION*nrowsPack*sizeof(Goldilocks::Element));
        hipMemcpy(challenges_d, challenges, params.challenges.degree()*FIELD_EXTENSION*nrowsPack*sizeof(Goldilocks::Element), hipMemcpyHostToDevice);
        
        hipMalloc(&challenges_ops_d, params.challenges.degree()*FIELD_EXTENSION*nrowsPack*sizeof(Goldilocks::Element));
        hipMemcpy(challenges_ops_d, challenges_ops, params.challenges.degree()*FIELD_EXTENSION*nrowsPack*sizeof(Goldilocks::Element), hipMemcpyHostToDevice);
        
        uint64_t *numbers = &parserArgs.numbers[parserParams.numbersOffset];
        Goldilocks::Element numbers_[parserParams.nNumbers*nrowsPack];
        for(uint64_t i = 0; i < parserParams.nNumbers; ++i) {
            for(uint64_t j = 0; j < nrowsPack; ++j) {
                numbers_[i*nrowsPack + j] = Goldilocks::fromU64(numbers[i]);
            }
        }
        hipMalloc(&numbers_d, parserParams.nNumbers*nrowsPack*sizeof(Goldilocks::Element));
        hipMemcpy(numbers_d, numbers_, parserParams.nNumbers*nrowsPack*sizeof(Goldilocks::Element), hipMemcpyHostToDevice);

        Goldilocks::Element publics[starkInfo.nPublics*nrowsPack];
        for(uint64_t i = 0; i < starkInfo.nPublics; ++i) {
            for(uint64_t j = 0; j < nrowsPack; ++j) {
                publics[i*nrowsPack + j] = params.publicInputs[i];
            }
        }

        hipMalloc(&publics_d, starkInfo.nPublics*nrowsPack*sizeof(Goldilocks::Element));
        hipMemcpy(publics_d, publics, starkInfo.nPublics*nrowsPack*sizeof(Goldilocks::Element), hipMemcpyHostToDevice);

        Goldilocks::Element evals[params.evals.degree()*FIELD_EXTENSION*nrowsPack];
        for(uint64_t i = 0; i < params.evals.degree(); ++i) {
            for(uint64_t j = 0; j < nrowsPack; ++j) {
                evals[(i*FIELD_EXTENSION)*nrowsPack + j] = params.evals[i][0];
                evals[(i*FIELD_EXTENSION + 1)*nrowsPack + j] = params.evals[i][1];
                evals[(i*FIELD_EXTENSION + 2)*nrowsPack + j] = params.evals[i][2];
            }
        }
        hipMalloc(&evals_d, params.evals.degree()*FIELD_EXTENSION*nrowsPack*sizeof(Goldilocks::Element));
        hipMemcpy(evals_d, evals, params.evals.degree()*FIELD_EXTENSION*nrowsPack*sizeof(Goldilocks::Element), hipMemcpyHostToDevice);

        /* 
            buffered data
        */
        hipMalloc(&constPols_d, params.pConstPols->numPols()*params.pConstPols->degree()*sizeof(Goldilocks::Element));
        hipMemcpy(constPols_d, params.pConstPols->address(), params.pConstPols->size(), hipMemcpyHostToDevice);
        
        hipMalloc(&constPols2ns_d, params.pConstPols2ns->numPols()*params.pConstPols2ns->degree()*sizeof(Goldilocks::Element));
        hipMemcpy(constPols2ns_d, params.pConstPols2ns->address(), params.pConstPols2ns->size(), hipMemcpyHostToDevice);

        hipMalloc(&x_d, params.x_n.dim()*params.x_n.degree()*sizeof(Goldilocks::Element));
        hipMemcpy(x_d, params.x_n.address(), params.x_n.size(), hipMemcpyHostToDevice);

        hipMalloc(&x_2ns_d, params.x_2ns.dim()*params.x_2ns.degree()*sizeof(Goldilocks::Element));
        hipMemcpy(x_2ns_d, params.x_2ns.address(), params.x_2ns.size(), hipMemcpyHostToDevice);

        hipMalloc(&zi_d, params.zi.dim()*params.zi.degree()*sizeof(Goldilocks::Element));
        hipMemcpy(zi_d, params.zi.address(), params.zi.size(), hipMemcpyHostToDevice);

        hipMalloc(&xDivXSubXi_d, params.xDivXSubXi.dim()*params.xDivXSubXi.degree()*sizeof(Goldilocks::Element));
        hipMemcpy(xDivXSubXi_d, params.xDivXSubXi.address(), params.xDivXSubXi.size(), hipMemcpyHostToDevice);

        hipMalloc(&pols_d, starkInfo.mapTotalN*sizeof(Goldilocks::Element));
        hipMemcpy(pols_d, params.pols, starkInfo.mapTotalN*sizeof(Goldilocks::Element), hipMemcpyHostToDevice);

        /*
            temporal buffers
        */
        hipMalloc(&bufferT_d, numBlocks*sizeof(Goldilocks::Element*));
        for(uint64_t i = 0; i < numBlocks; ++i) {
            hipMalloc(&bufferT_d[i], 2*nCols*nrowsPack*sizeof(Goldilocks::Element));
        }
        hipMalloc(&tmp1_d, numBlocks*sizeof(Goldilocks::Element*));
        for(uint64_t i = 0; i < numBlocks; ++i) {
            hipMalloc(&tmp1_d[i], parserParams.nTemp1*nrowsPack*sizeof(Goldilocks::Element));
        }
        hipMalloc(&tmp3_d, numBlocks*sizeof(Goldilocks::Element*));
        for(uint64_t i = 0; i < numBlocks; ++i) {
            hipMalloc(&tmp3_d[i], parserParams.nTemp3*nrowsPack*FIELD_EXTENSION*sizeof(Goldilocks::Element));
        }
}    
